#include "hip/hip_runtime.h"
#include "opencv2/core.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/imgproc.hpp"
#include "textures.h"

hipTextureObject_t poca_mus::AddTexByFile(std::string file_path) {
    cv::Mat src = cv::imread(file_path);
    cv::Mat dst;
    cv::cvtColor(src, dst, CV_BGR2RGBA);

    int width = dst.cols;
    int height = dst.rows;

    hipArray* cu_array;
    hipChannelFormatDesc cu_desc = hipCreateChannelDesc<uchar4>();
    checkCudaErrors(hipMallocArray(&cu_array, &cu_desc, width, height));
    checkCudaErrors(hipMemcpy2DToArray(cu_array, 0, 0, dst.data, width * 4, width * sizeof(uint8_t), height,
                                        hipMemcpyHostToDevice));

    hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = cu_array;

    hipTextureDesc tex_desc;
    tex_desc.addressMode[0] = hipAddressModeMirror;
    tex_desc.addressMode[1] = hipAddressModeMirror;
    tex_desc.filterMode = hipFilterModeLinear;
    tex_desc.readMode = hipReadModeNormalizedFloat;
    tex_desc.normalizedCoords = 1;

    hipTextureObject_t tex_object;

    checkCudaErrors(hipCreateTextureObject(&tex_object, &res_desc, &tex_desc, NULL));
    return tex_object;
}

__device__ Float4 poca_mus::GetTex2D(hipTextureObject_t tex_obj, float u, float v) {
    float4 rgb = tex2D<float4>(tex_obj, u, v);
    return Float4(rgb.x, rgb.y, rgb.z, 1.f);
}
